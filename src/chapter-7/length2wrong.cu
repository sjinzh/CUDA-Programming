
#include <hip/hip_runtime.h>
#include <stdlib.h> // malloc() and free()
#include <stdio.h> // printf()
#include <math.h> // sqrt()
#include <time.h> // clock_t, clock(), and CLOCKS_PER_SEC
double get_length(double *x, int N);

int main(void)
{
    int N = 1000;
    int M = sizeof(double) * N;
    double *x = (double *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
    }
    double length = get_length(x, N);
    printf("length = %g.\n", length);
    free(x);
    return 0;
}

void __global__ get_length(double *g_x, double *g_length, int N)
{
    int tid = threadIdx.x;
    if (tid < N)
    {
        g_x[tid] *= g_x[tid];
    }
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if (tid < offset)
        {
            if (tid + offset < N)
            {
                g_x[tid] += g_x[tid + offset];
            }
        }
    }
    if (tid == 0)
    {
        g_length[0] = sqrt(g_x[0]);
    }
}

double get_length(double *x, int N)
{
    double *g_length;
    hipMalloc((void**)&g_length, sizeof(double));
    double *g_x;
    hipMalloc((void**)&g_x, sizeof(double) * N);
    hipMemcpy(g_x, x, sizeof(double) * N, hipMemcpyHostToDevice);
    for (int n = 0; n < 100000; n++)
    {
        hipMemcpy(g_x, x, sizeof(double) * N, hipMemcpyHostToDevice);
        get_length<<<1, 1024>>>(g_x, g_length, N);
    }
    double *cpu_length = (double *) malloc(sizeof(double));
    hipMemcpy(cpu_length, g_length, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(g_length);
    hipFree(g_x);
    double length = cpu_length[0];
    free(cpu_length);
    return length;
}
